#include "hip/hip_runtime.h"
#include "Obj_cuda.hpp"
#include <stdio.h>
#include <boost/shared_ptr.hpp>
#include <boost/make_shared.hpp>

__global__ void compute(int* res, int* a, int* b)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
res[i] = 10 *(a[i] + b[i]);
}

int Obj_cuda::operator()(int* res, int* a, int* b ,int N)
{
boost::shared_ptr<bool> ready_;
compute<<<1,N>>>(res,a,b);
return 1;
}

